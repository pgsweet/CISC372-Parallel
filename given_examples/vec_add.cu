
#include <hip/hip_runtime.h>
#include <stdio.h>

void Print_vector(const char title[], const float x[], const int n) {
   printf("%s = ", title);
   for (int i = 0; i < n; i++)
      printf("%.2f ", x[i]);
   printf("\n");
}  /* Print_vector */

void Rand_vector(float x[], const int n) {
   for (int i = 0; i < n; i++)
      x[i] = random()/((double) RAND_MAX);
}  /* Rand_vector */

void Vec_add(const float a[], const float b[], float c[], const int n) {
    for (int i=0; i<n; i++) {
        c[i] = a[i] + b[i];
    }
} /* Vec_add */

int main(int argc, char *argv[]) {
    int blk_ct, th_per_blk;
    int n;

    if (argc != 4) {
        fprintf(stderr, "usage: %s <n> <blk_ct> <th_per_blk>\n", argv[0]);
        exit(0);
    } else {
        n = strtol(argv[1], NULL, 10);
        blk_ct = strtol(argv[2], NULL, 10);
        th_per_blk = strtol(argv[3], NULL, 10);
    }

    float *a = (float*)malloc(n*sizeof(float));
    float *b = (float*)malloc(n*sizeof(float));
    float *c = (float*)malloc(n*sizeof(float));
    
    srandom(12345);
    Rand_vector(a, n);
    Rand_vector(b, n);
    Rand_vector(c, n);

    Vec_add(a, b, c, n);

    Print_vector("a", a, n);
    Print_vector("b", b, n);
    Print_vector("c", c, n);

    free(a);
    free(b);
    free(c);

    return 0;
} /* main */